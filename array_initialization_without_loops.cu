#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 1024

__global__ void assign_zero(int *a)
{
    int id = threadIdx.x;
    a[id] = 0;
}

int main()
{
    int a[N], *a_d;
    for(int i=0; i<5; i++)
    {
        printf("%d ", a[i]);
    }
    printf("\n");
    hipMalloc((void**) &a_d, N*sizeof(int));

    assign_zero<<<1, N>>>(a_d);

    hipMemcpy(a, a_d, N*sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i=0; i<5; i++)
    {
        printf("%d ", a[i]);
    }
    return 0;

}

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add (int n, float* a, float* b, float* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = (a[i] + b[i]) / (a[i]*b[i]);
    } 
}

int main()
{
    int N = 607374182;
    int BLOCK_SIZE = 256;
    float *a, *b, *c;

    // The following malloc is to use pinned memory. If you want to allocate pagable memory, you can just use regular malloc().
    hipHostMalloc((void**) &a, N*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &b, N*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**) &c, N*sizeof(float), hipHostMallocDefault);

    for (int i = 0; i< N; i++)
    {
        a[i] = i;
        b[i] = 2*i;
    }

    float *a_d, *b_d, *c_d;

    hipMalloc((void**) &a_d, N*sizeof(float));
    hipMalloc((void**) &b_d, N*sizeof(float));
    hipMalloc((void**) &c_d, N*sizeof(float));

    hipMemcpy(a_d, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, N*sizeof(float), hipMemcpyHostToDevice);
    
    add<<<ceil(N/(float)BLOCK_SIZE), BLOCK_SIZE>>>(N, a_d, b_d, c_d);

    hipMemcpy(c, c_d, N*sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i< 10; i++)
    {
        std::cout<<a[i]<< " "<<b[i]<<" "<<c[i] << std::endl;
    }
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    
    return 0;
}